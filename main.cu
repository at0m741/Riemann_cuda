#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   main.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: at0m <at0m@student.42.fr>                  +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/12/05 13:08:37 by at0m              #+#    #+#             */
/*   Updated: 2023/12/05 13:15:29 by at0m             ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <cstdio>
#include "hip/hip_runtime.h"
#include  <math.h>
#include "lol.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <GL/glut.h>

__global__
void GPUFunction_zeta(double s_r, double s_i, double *real, double *imag)
{
	int	n;
	double a;
	double b;

	n = 1;
	printf("zeta arg : %f + i * %f\n", s_r, s_i);
	while (n < 5000)
	{
		a = cos(s_i * log(n));
		b = sin(s_i * log(n));
		*real += a / ((pow(a, 2) + pow(b, 2)) * pow(n, s_r));
		*imag += -b / ((pow(a, 2) + pow(b, 2)) * pow(n, s_r));
		n++;
	}
}

__global__
void	riemann(t_var *f, t_riemann *r, double t, double sigma)
{
	double			tmp;
	unsigned int	n;
	double			max;
	int				ntz = 0;

	n = 0;
	max = 7500;
	GPUFunction_zeta<<<1, 12>>>(sigma, t, &(f->c_r), &(f->c_i));
	if (approx(sigma) && approx2(f->c_i) && approx2(f->c_r))
	{
		ntz = 1;
		printf("NTZ\n");
	}
	f->c_r = (r->w1 * sign(f->c_r) * log(ft_dabs(f->c_r)) + r->w2);
	f->c_i = (r->w3 * sign(f->c_i) * log(ft_dabs(f->c_i)) + r->w4);
	while (module(f->z_r, f->z_i) <= 2 && n < max)
	{
		tmp = f->z_r;
		f->z_r = (f->z_r * f->z_r) - f->z_i * f->z_i + f->c_r;
		f->z_i = 2 * tmp * f->z_i + f->c_i;
		n++;
	}
	if (n == max)
		return ;
}

__global__
void GPUFunction()
{
  printf("hello from the Gpu.\n");
}

int main()
{
	//unsigned int	p;
	t_var			    *f_mandelbrodt;
	t_riemann		  *f_r;
	


  f_r = (t_riemann *)malloc(sizeof(t_riemann));
  f_r->smin = -14;
  f_r->smax = 8;
  f_r->tmin = -16;
  f_r->tmax = 16;
  if (f_r->smax - f_r->smin != 0)
  {
    f_r->h = floor(900 *  (f_r->tmax - f_r->tmin) / (f_r->smax - f_r->smin));
    f_r->w1 = 2.47 / (f_r->smax - f_r->smin);
    f_r->w2 = (0.47 * f_r->smin + 2 * f_r->smax) / (f_r->smax - f_r->smin);
    f_r->w3 = 2.24 / (f_r->tmax - f_r->tmin);
    f_r->w4 = 1.12 * (f_r->tmin + f_r->tmax) / (f_r->tmin - f_r->tmax);
  }
			//double sigma = f_r->smin + x * (f_r->smax - f_r->smin) / (WIDTH);
			//p = riemann(ft_init_var_mandelbrodt(), f_r, t, sigma);
			//p = julia(x, y, f_julia, 1);

  GPUFunction<<<1, 1>>>();

  for (int y = 0; y < 900; y++)
	{
		double t = f_r->tmin + y * (f_r->tmax - f_r->tmin) / (900);
		for (int x = 0; x < 900; x++)
		{
			double sigma = f_r->smin + x * (f_r->smax - f_r->smin) / (900);
		
			riemann<<<1, 100>>>(f_mandelbrodt, f_r, t, sigma);
			std::cout << "x : " << x << " y : " << y << std::endl;
		}
	}

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}