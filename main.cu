#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   main.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: at0m <at0m@student.42.fr>                  +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/12/05 13:08:37 by at0m              #+#    #+#             */
/*   Updated: 2023/12/05 13:15:29 by at0m             ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "lol.h"

__global__
void GPUFunction_zeta(double s_r, double s_i, double *real, double *imag)
{
	int	n;
	double a;
	double b;

	n = 1;
	printf("zeta arg : %f + i * %f\n", s_r, s_i);
	while (n < 5000)
	{
		a = cos(s_i * log(n));
		b = sin(s_i * log(n));
		*real += a / ((pow(a, 2) + pow(b, 2)) * pow(n, s_r));
		*imag += -b / ((pow(a, 2) + pow(b, 2)) * pow(n, s_r));
		n++;
	}
}

__global__
void	riemann(t_var *f, t_riemann *r, double t, double sigma, unsigned int *color)
{
	double			tmp;
	unsigned int	n;
	double			max;
	int				ntz = 0;

	n = 0;
	max = 7500;
	GPUFunction_zeta<<<16, 16>>>(sigma, t, &(f->c_r), &(f->c_i));
	if (approx(sigma) && approx2(f->c_i) && approx2(f->c_r))
	{
		ntz = 1;
		printf("NTZ\n");
	}
	f->c_r = (r->w1 * sign(f->c_r) * log(ft_dabs(f->c_r)) + r->w2);
	f->c_i = (r->w3 * sign(f->c_i) * log(ft_dabs(f->c_i)) + r->w4);
	while (module(f->z_r, f->z_i) <= 2 && n < max)
	{
		tmp = f->z_r;
		f->z_r = (f->z_r * f->z_r) - f->z_i * f->z_i + f->c_r;
		f->z_i = 2 * tmp * f->z_i + f->c_i;
		n++;
	}
	printf("CEST LE n : %d\n", n);
    if (n == max) {
		printf("max\n");
        *color = 0x00000000; // Noir pour max atteint
    } else {
        *color = 0x00FFFFFF; // Blanc sinon
    }
}

__global__
void GPUFunction()
{
  printf("hello from the Gpu.\n");
}

int main()
{
	//unsigned int	p;
	void				*mlx;
	void				*win;
	struct s_frame		f;
	t_var			    *f_mandelbrodt;
	t_riemann		  	*f_r;
	
	mlx = mlx_init();
	win = mlx_new_window(mlx, WIDTH, HEIGHT, "fract-ol");
	f.img = mlx_new_image(mlx, WIDTH, HEIGHT);
  	f_r = (t_riemann *)malloc(sizeof(t_riemann));
  	f_r->smin = -14;
  	f_r->smax = 8;
  	f_r->tmin = -16;
  	f_r->tmax = 16;

  	f_r->h = floor(900 *  (f_r->tmax - f_r->tmin) / (f_r->smax - f_r->smin));
  	f_r->w1 = 2.47 / (f_r->smax - f_r->smin);
  	f_r->w2 = (0.47 * f_r->smin + 2 * f_r->smax) / (f_r->smax - f_r->smin);
  	f_r->w3 = 2.24 / (f_r->tmax - f_r->tmin);
  	f_r->w4 = 1.12 * (f_r->tmin + f_r->tmax) / (f_r->tmin - f_r->tmax);

  	for (int y = 0; y < 900; y++)
	{
		double t = f_r->tmin + y * (f_r->tmax - f_r->tmin) / (900);
		for (int x = 0; x < 900; x++)
		{
			double sigma = f_r->smin + x * (f_r->smax - f_r->smin) / (900);
			unsigned int color = 0;
			//printf("x : %d, y : %d, color : %d\r", x, y, color);
        	riemann<<<16, 16>>>(f_mandelbrodt, f_r, t, sigma, &color);
			//printf("x : %d, y : %d, color : %d\r", x, y, color);
        	mlx_pixel_put(mlx, win, x, y, color);
		}
	}
  	hipDeviceSynchronize();
	mlx_put_image_to_window(mlx, win, f.img, 0, 0);
	mlx_loop(mlx);

  	return EXIT_SUCCESS;
}