#include "hip/hip_runtime.h"

#include "lol.h"


__host__ __device__
double	ft_dabs(double n)
{
	if (n < 0)
		return (-n);
	return (n);
}
__host__ __device__
int	sign(double n)
{
	if (n < 0)
		return (-1);
	return (1);
}
__host__ __device__
double		approx(double n)
{
	return (n > 0.495 && n < 0.505);
}
__host__ __device__
double		approx2(double n)
{
	return (n > -0.5 && n < 0.5);
}
__host__ __device__
double	module(double a, double b)
{
	return (sqrt((a * a) + (b * b)));
}